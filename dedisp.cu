#include "hip/hip_runtime.h"
#include "dedisp.h"
#include "math.h"

//Tile properties, size of each tile is 48KB,
//if your shared memory is smaller, make these smaller
#define TILE_WIDTH_F 16
#define TILE_WIDTH_T 768
#define WARP 32

__device__ __constant__ float c_DMs[1024];

//For f in GHz, DM in pc*cm^-3
#define OFFSET(f,DM) roundf((4.149/(f)/(f)*(DM))/(dt))
#define RELATIVE_OFFSET(ch) roundf((OFFSET((f_ctr-df*((numchan-1)/2.0-(ch)-0*TILE_WIDTH_F)),DM) - OFFSET(f_ctr+(numchan-1)/2.0*df,DM)))


//The data is parallelised in the way that each block has every DM and covers few channels,
//while each thread in a single block has a particular DM.
//It seems that one block can hold only 1024 thread, so USE NO MORE THAN 1024 numDMs
//Each thread goes through all t and sum their channels to add to global accumulator


__global__ void setComplexZero(hipfftComplex* d_dest, int arraysize){
	hiprandState_t state;
	hiprand_init(threadIdx.x*threadIdx.x,threadIdx.x,threadIdx.x,&state);
	for(int i=0; i<arraysize; i++)	*(d_dest+i+arraysize*threadIdx.x) = (hipfftComplex){2*hiprand_normal(&state)+10,0};
}

__global__ void timeshiftKernel(float* d_input, hipfftComplex* d_output, float f_ctr, float df, 
				int tsize, int numchan, float dt){
	
	__device__ __shared__ float sharedInput[TILE_WIDTH_F][TILE_WIDTH_T];
	float DM = c_DMs[threadIdx.x];
	//The relative offset is quite small(<10000) so that every two of
	//them is forced to be stored in one 32-bits register to reduce the use
	//of registers.
	int off_0_1 = 65536* RELATIVE_OFFSET(0) + RELATIVE_OFFSET(1);
	int off_2_3 = 65536* RELATIVE_OFFSET(2) + RELATIVE_OFFSET(3);
	int off_4_5 = 65536* RELATIVE_OFFSET(4) + RELATIVE_OFFSET(5);
	int off_6_7 = 65536* RELATIVE_OFFSET(6) + RELATIVE_OFFSET(7);
	int off_8_9 = 65536* RELATIVE_OFFSET(8) + RELATIVE_OFFSET(9);
	int off_10_11 = 65536* RELATIVE_OFFSET(10) + RELATIVE_OFFSET(11);
	int off_12_13 = 65536* RELATIVE_OFFSET(12) + RELATIVE_OFFSET(13);
	int off_14_15 = 65536* RELATIVE_OFFSET(14) + RELATIVE_OFFSET(15);
	for ( int t=tsize; t>=TILE_WIDTH_T; t-=TILE_WIDTH_T ){
		
		__syncthreads();
		///Note that only the first 32 threads (first warp) is used to load,
		///so make sure your data RUN AT LEAST 32 DMs
		if(threadIdx.x<WARP)
			for ( int i=0; i<TILE_WIDTH_T*TILE_WIDTH_F/WARP; i++ )	sharedInput[0][i*WARP+threadIdx.x] = 
				*(d_input+(blockIdx.x*TILE_WIDTH_F + (i*WARP+threadIdx.x)/TILE_WIDTH_T)*tsize 
 					+ ((i*WARP+threadIdx.x)%TILE_WIDTH_T ) + (t-TILE_WIDTH_T));
		__syncthreads();
		
		for ( int tj=0; tj<TILE_WIDTH_T; tj++ ) {

			//This loop has been manually broken down to make sure that
			//all the offsets are stored in registers, for once you use a
			//loop, the compiler won't be able to determine the addresses
			//at compile time so that the offsets must be settled in global
			//memory and the load will be much slower. These code might be
			//extremely ugly and hard to maintain but it's very important.

			//Only write if the all can be write to correct position
			if(t-TILE_WIDTH_T+tj-off_0_1/65536>0){
			atomicAdd((float*)(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_0_1/65536) ,sharedInput[0][tj]);
			atomicAdd((float*)(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_0_1%65536) ,sharedInput[1][tj]);
			atomicAdd((float*)(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_2_3/65536) ,sharedInput[2][tj]);
			atomicAdd((float*)(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_2_3%65536) ,sharedInput[3][tj]);
			atomicAdd((float*)(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_4_5/65536) ,sharedInput[4][tj]);
			atomicAdd((float*)(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_4_5%65536) ,sharedInput[5][tj]);
			atomicAdd((float*)(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_6_7/65536) ,sharedInput[6][tj]);
			atomicAdd((float*)(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_6_7%65536) ,sharedInput[7][tj]);
			atomicAdd((float*)(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_8_9/65536) ,sharedInput[8][tj]);
			atomicAdd((float*)(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_8_9%65536) ,sharedInput[9][tj]);
			atomicAdd((float*)(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_10_11/65536) ,sharedInput[10][tj]);
			atomicAdd((float*)(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_10_11%65536) ,sharedInput[11][tj]);
			atomicAdd((float*)(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_12_13/65536) ,sharedInput[12][tj]);
			atomicAdd((float*)(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_12_13%65536) ,sharedInput[13][tj]);
			atomicAdd((float*)(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_14_15/65536) ,sharedInput[14][tj]);
			atomicAdd((float*)(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_14_15%65536) ,sharedInput[15][tj]);

			//Note that atomicAdd may hinder performance, especially on old cards
			//However, race conditions only happens between different BLOCKS,
			//so if you are running only one block at any time, use normal add.
			//And even you're running not too many (4 or 5) blocks, the chance
			//of race conditions happening is quite negligible.
			//PS: When you are feeling lucky, use normal add:

			
			/*(*(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_0_1/65536)).x+=sharedInput[0][tj+0*TILE_WIDTH_T];
			(*(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_0_1%65536)).x+=sharedInput[0][tj+1*TILE_WIDTH_T];
			(*(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_2_3/65536)).x+=sharedInput[0][tj+2*TILE_WIDTH_T];
			(*(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_2_3%65536)).x+=sharedInput[0][tj+3*TILE_WIDTH_T];
			(*(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_4_5/65536)).x+=sharedInput[0][tj+4*TILE_WIDTH_T];
			(*(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_4_5%65536)).x+=sharedInput[0][tj+5*TILE_WIDTH_T];
			(*(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_6_7/65536)).x+=sharedInput[0][tj+6*TILE_WIDTH_T];
			(*(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_6_7%65536)).x+=sharedInput[0][tj+7*TILE_WIDTH_T];
			(*(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_8_9/65536)).x+=sharedInput[0][tj+8*TILE_WIDTH_T];
			(*(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_8_9%65536)).x+=sharedInput[0][tj+9*TILE_WIDTH_T];
			(*(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_10_11/65536)).x+=sharedInput[0][tj+10*TILE_WIDTH_T];
			(*(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_10_11%65536)).x+=sharedInput[0][tj+11*TILE_WIDTH_T];
			(*(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_12_13/65536)).x+=sharedInput[0][tj+12*TILE_WIDTH_T];
			(*(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_12_13%65536)).x+=sharedInput[0][tj+13*TILE_WIDTH_T];
			(*(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_14_15/65536)).x+=sharedInput[0][tj+14*TILE_WIDTH_T];
			(*(d_output+tsize*threadIdx.x+t-TILE_WIDTH_T+tj-off_14_15%65536)).x+=sharedInput[0][tj+15*TILE_WIDTH_T];*/
			
			}
		}
	};
}

void dedispersion(float* f_t, int numchan, int tsize,
		  float f_ctr, float df, float dt, float* DMs, int numDMs, float* output_dm_t, float* output_f_t){

	//Warnings
	if(tsize%TILE_WIDTH_T) printf("Warning: tsize is not a multiple of TILE_WIDTH_T, some data will be ignored\n");
	if(numchan%TILE_WIDTH_F) printf("Warning: numchan is not a multiple of TILE_WIDTH_F, some data will be ignored\n");
	if(numDMs<32) {printf("Error: numDMs is less than 32, it leads to serious error\n"); exit(1);}
	if(DMs==NULL) {printf("Error: DMs are not specified!\n"); exit(1);}

	//Write to __constant__ memory
	int sizeofDMArray = numDMs*sizeof(float);
	hipMemcpyToSymbol(HIP_SYMBOL(c_DMs),DMs,sizeofDMArray,0,hipMemcpyHostToDevice);

	int input_size = numchan*tsize*sizeof(float);
	int output_size = numDMs*tsize*sizeof(hipfftComplex);
	if(input_size+output_size>=2147483648) {printf("Error:Your data is too big!\n"); exit(1);}

	//Allocate device memory for input and output
	float* d_input;				//input
	hipfftComplex* d_output;			//output
	hipMalloc((void**)&d_input,input_size);
	hipMalloc((void**)&d_output,output_size);
	//Copy host memory to device memory
	hipMemcpy(d_input,f_t,input_size,hipMemcpyHostToDevice);

	//Launch a small kernel to initialize output array
	dim3 dimBlock(numDMs,1,1);
	setComplexZero<<<1,dimBlock>>>(d_output,tsize);

	//Launch main kernel to do time shift
	dim3 dimGrid(numchan/TILE_WIDTH_F,1,1);
	timeshiftKernel<<<dimGrid,dimBlock>>>(d_input,d_output,f_ctr,df,tsize,numchan,dt);

	//Copy output from device memory to host memory
	hipFree(d_input);
	hipMemcpy((void*)output_dm_t,(void*)d_output,output_size,hipMemcpyDeviceToHost);
	
	//Do FFT
	hipfftHandle plan;
	int n[1]={tsize};
	hipfftPlanMany(&plan,1,n,n,1,tsize,n,1,tsize,HIPFFT_C2C,numDMs);
	
	hipfftExecC2C(plan,d_output,d_output,HIPFFT_FORWARD);
	hipfftDestroy(plan);

	hipMemcpy(output_f_t,d_output,output_size,hipMemcpyDeviceToHost);
	//Write Z(DM,f) to files
	
	hipFree(d_output);
}

int main(){

	//Initialize observation
	int numchan = 64*TILE_WIDTH_F;
	int numsignal = 256;
	int tsize = TILE_WIDTH_T*numsignal;
	float f_ctr = 17.5;
	float df = 0.02;
	int numDMs = 256;
	float dt = 0.001;

	//Make output file names

	FILE* fp=fopen("DM_t.txt","wb");
	FILE* fp2=fopen("DM_f.txt","wb");
	FILE* fp3=fopen("f_t.txt","wb");

	int k;
	//Initialize DM array
	float *DMs;
	DMs = (float*)malloc(numDMs*sizeof(float));
	int i=0;
	for (i=0;i<numDMs;i++) *(DMs+i)=i*1.0;
	
	//Initialize input array(Fake data)
	float* f_t=NULL;
	float fakeDM=120;
	f_t = (float*)malloc(sizeof(float)*numchan*tsize);
	for ( k=0;k<numchan*tsize;k++)
		*(f_t+k)=0.0f;
	printf("Set to zero...\n");
	for ( k=0;k<numchan;k++){
		float f=f_ctr-((numchan-1)/2.0-k)*df;
		for ( i=0 ; i<numsignal; i++){
		*(f_t+k*tsize+200+i*768+(int)round(4.149/f/f*fakeDM/dt))=100;
		*(f_t+k*tsize+200+i*768+(int)round(4.149/f/f*fakeDM/dt)-1)=69;
		*(f_t+k*tsize+200+i*768+(int)round(4.149/f/f*fakeDM/dt)+1)=69;
		*(f_t+k*tsize+200+i*768+(int)round(4.149/f/f*fakeDM/dt)-2)=37;
		*(f_t+k*tsize+200+i*768+(int)round(4.149/f/f*fakeDM/dt)+2)=37;
		*(f_t+k*tsize+200+i*768+(int)round(4.149/f/f*fakeDM/dt)-3)=18;
		*(f_t+k*tsize+200+i*768+(int)round(4.149/f/f*fakeDM/dt)+3)=18;
		*(f_t+k*tsize+200+i*768+(int)round(4.149/f/f*fakeDM/dt)-4)=6;
		*(f_t+k*tsize+200+i*768+(int)round(4.149/f/f*fakeDM/dt)+4)=6;
		}
	}

	float *t_output,*f_output;
	t_output = (float*)malloc(2*tsize*numDMs*sizeof(float));
	f_output = (float*)malloc(2*tsize*numDMs*sizeof(float));
	

	printf("\nStart de-dispersion!\n");	
	dedispersion(f_t, numchan, tsize, f_ctr, df ,dt, DMs, numDMs , t_output, f_output);

	//Print output
	for (i=0; i<numchan*tsize; i++)
	fwrite(f_t+i,4,1,fp3);
	printf("Input wrote to f_t.txt, in %d*%d float\n",numchan,tsize);
	for (i=0; i<numDMs*tsize; i++)
	fwrite(t_output+2*i,4,1,fp);
	printf("Time series wrote to DM_t.txt, in %d*%d float\n",numDMs,tsize);
	for (i=0; i<numDMs*tsize; i++){
		float p = f_output[2*i]*f_output[2*i] + f_output[2*i+1]*f_output[2*i+1];
		fwrite(&p,4,1,fp2);
	}
	printf("Frequency series wrote to DM_f.txt, in %d*%d float\n",numDMs,tsize);
	fclose(fp);
	fclose(fp2);
	fclose(fp3);
}
